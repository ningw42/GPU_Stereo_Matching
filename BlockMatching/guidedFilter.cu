#include "guidedFilter.cuh"
#include "KernelFunction.cuh"

guidedFilterGPU::guidedFilterGPU(int _rows, int _cols, int _r, int _c, float _eps)
{
	rows = _rows;
	cols = _cols;
	r = _r;
	c = _c;
	eps = _eps;
	total = rows * cols;
	// temp 
	hipMalloc(&I, total * sizeof(float));
	hipMalloc(&p, total * sizeof(float));
	hipMalloc(&sq_I, total * sizeof(float));
	hipMalloc(&sq_mean_I, total * sizeof(float));
	hipMalloc(&mul_Ip, total * sizeof(float));
	hipMalloc(&mul_mean_Ip_mean_p, total * sizeof(float));
	hipMalloc(&sum_varI_eps, total * sizeof(float));
	hipMalloc(&mul_a_meanI, total * sizeof(float));

	hipMalloc(&mul_meana_I, total * sizeof(float));
	hipMalloc(&result_float, total * sizeof(float));
	// useful
	hipMalloc(&mean_I, total * sizeof(float));
	hipMalloc(&mean_II, total * sizeof(float));
	hipMalloc(&mean_p, total * sizeof(float));
	hipMalloc(&var_I, total * sizeof(float));
	hipMalloc(&mean_Ip, total * sizeof(float));
	hipMalloc(&cov_Ip, total * sizeof(float));
	hipMalloc(&a, total * sizeof(float));
	hipMalloc(&b, total * sizeof(float));
	hipMalloc(&mean_a, total * sizeof(float));
	hipMalloc(&mean_b, total * sizeof(float));
}

guidedFilterGPU::guidedFilterGPU()
{

}

guidedFilterGPU::~guidedFilterGPU()
{
}

void guidedFilterGPU::filter(uchar *I_uchar, uchar *p_uchar, uchar *result)
{
	kernelConvertToFloat << <rows, cols >> >(I_uchar, I); // I convert to float
	kernelConvertToFloat << <rows, cols >> >(p_uchar, p); // p convert to float

	kernelBoxFilter << <rows, cols >> >(I, mean_I, r, c, rows, cols); // mean_I = boxfilter(I, r, c)

	kernelMul << <rows, cols >> >(I, I, sq_I); // 
	kernelBoxFilter << <rows, cols >> >(sq_I, mean_II, r, c, rows, cols);// mean_II = boxfilter(I.mul(I), r, c)

	kernelMul << <rows, cols >> >(mean_I, mean_I, sq_mean_I); // can be optimized
	kernelSub << <rows, cols >> >(mean_II, sq_mean_I, var_I); // var_I = mean_II - mean_I.mul(mean_I)

	kernelBoxFilter << <rows, cols >> >(p, mean_p, r, c, rows, cols); // mean_p = boxfilter(p, r, c)
	
	kernelMul << <rows, cols >> >(I, p, mul_Ip); // 
	kernelBoxFilter << <rows, cols >> >(mul_Ip, mean_Ip, r, c, rows, cols); // mean_Ip = boxfilter(I.mul(p), r, c)

	kernelMul << <rows, cols >> >(mean_I, mean_p, mul_mean_Ip_mean_p); // can be optimized
	kernelSub << <rows, cols >> >(mean_Ip, mul_mean_Ip_mean_p, cov_Ip); // cov_Ip = mean_Ip - mean_I.mul(mean_p);

	kernelAddEle << <rows, cols >> >(var_I, eps, sum_varI_eps); // can be optimized
	kernelDivide << <rows, cols >> >(cov_Ip, sum_varI_eps, a); // a = cov_Ip / (var_I + eps)

	kernelMul << <rows, cols >> >(a, mean_I, mul_a_meanI); // can be optimized
	kernelSub << <rows, cols >> >(mean_p, mul_a_meanI, b); // b = mean_p - a.mul(mean_I);

	kernelBoxFilter << <rows, cols >> >(a, mean_a, r, c, rows, cols); // mean_a = boxfilter(a, r, c)

	kernelBoxFilter << <rows, cols >> >(b, mean_b, r, c, rows, cols); // mean_b = boxfilter(b, r, c)

	kernelMul << <rows, cols >> >(mean_a, I, mul_meana_I); // can be optimized
	kernelAdd << <rows, cols >> >(mul_meana_I, mean_b, result_float); // return mean_a.mul(I) + mean_b

	kernelConvertToUchar << <rows, cols >> >(result_float, result); // 
}
