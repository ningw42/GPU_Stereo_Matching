#include "guidedFilter.cuh"
#include "KernalFunction.cuh"

guidedFilterGPU::guidedFilterGPU(int _rows, int _cols, int _r, int _c, float _eps)
{
	rows = _rows;
	cols = _cols;
	r = _r;
	c = _c;
	eps = _eps;
	total = rows * cols;
	// temp 
	hipMalloc(&I, total * sizeof(float));
	hipMalloc(&p, total * sizeof(float));
	hipMalloc(&sq_I, total * sizeof(float));
	hipMalloc(&sq_mean_I, total * sizeof(float));
	hipMalloc(&mul_Ip, total * sizeof(float));
	hipMalloc(&mul_mean_Ip_mean_p, total * sizeof(float));
	hipMalloc(&sum_varI_eps, total * sizeof(float));
	hipMalloc(&mul_a_meanI, total * sizeof(float));

	hipMalloc(&mul_meana_I, total * sizeof(float));
	hipMalloc(&result_float, total * sizeof(float));
	// useful
	hipMalloc(&mean_I, total * sizeof(float));
	hipMalloc(&mean_II, total * sizeof(float));
	hipMalloc(&mean_p, total * sizeof(float));
	hipMalloc(&var_I, total * sizeof(float));
	hipMalloc(&mean_Ip, total * sizeof(float));
	hipMalloc(&cov_Ip, total * sizeof(float));
	hipMalloc(&a, total * sizeof(float));
	hipMalloc(&b, total * sizeof(float));
	hipMalloc(&mean_a, total * sizeof(float));
	hipMalloc(&mean_b, total * sizeof(float));
}

guidedFilterGPU::guidedFilterGPU()
{

}

guidedFilterGPU::~guidedFilterGPU()
{
}

void guidedFilterGPU::filter(uchar *I_uchar, uchar *p_uchar, uchar *result)
{
	kernalConvertToFloat << <rows, cols >> >(I_uchar, I); // I convert to float
	kernalConvertToFloat << <rows, cols >> >(p_uchar, p); // p convert to float

	kernalBoxFilter << <rows, cols >> >(I, mean_I, r, c, rows, cols); // mean_I = boxfilter(I, r, c)

	kernalMul << <rows, cols >> >(I, I, sq_I); // 
	kernalBoxFilter << <rows, cols >> >(sq_I, mean_II, r, c, rows, cols);// mean_II = boxfilter(I.mul(I), r, c)

	kernalMul << <rows, cols >> >(mean_I, mean_I, sq_mean_I); //
	kernalSub << <rows, cols >> >(mean_II, sq_mean_I, var_I); // var_I = mean_II - mean_I.mul(mean_I)

	kernalBoxFilter << <rows, cols >> >(p, mean_p, r, c, rows, cols); // mean_p = boxfilter(p, r, c)
	
	kernalMul << <rows, cols >> >(I, p, mul_Ip); // 
	kernalBoxFilter << <rows, cols >> >(mul_Ip, mean_Ip, r, c, rows, cols); // mean_Ip = boxfilter(I.mul(p), r, c)

	kernalMul << <rows, cols >> >(mean_I, mean_p, mul_mean_Ip_mean_p);
	kernalSub << <rows, cols >> >(mean_Ip, mul_mean_Ip_mean_p, cov_Ip); // cov_Ip = mean_Ip - mean_I.mul(mean_p);

	kernalAddEle << <rows, cols >> >(var_I, eps, sum_varI_eps); //
	kernalDivide << <rows, cols >> >(cov_Ip, sum_varI_eps, a); // a = cov_Ip / (var_I + eps)

	kernalMul << <rows, cols >> >(a, mean_I, mul_a_meanI); //
	kernalSub << <rows, cols >> >(mean_p, mul_a_meanI, b); // b = mean_p - a.mul(mean_I);

	kernalBoxFilter << <rows, cols >> >(a, mean_a, r, c, rows, cols); // mean_a = boxfilter(a, r, c)

	kernalBoxFilter << <rows, cols >> >(b, mean_b, r, c, rows, cols); // mean_b = boxfilter(b, r, c)

	kernalMul << <rows, cols >> >(mean_a, I, mul_meana_I);
	kernalAdd << <rows, cols >> >(mul_meana_I, mean_b, result_float); // return mean_a.mul(I) + mean_b

	kernalConvertToUchar << <rows, cols >> >(result_float, result); // 
}
