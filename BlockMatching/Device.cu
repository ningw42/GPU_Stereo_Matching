#include "hip/hip_runtime.h"
#include "Device.cuh"
#include "BlockMatching.h"
#include "guidedfilter.h"
#include "KernelFunction.cuh"

using namespace std;
using namespace cv;




Device::Device(Size size, int numDisp, int wsz, Mat &mx1, Mat &my1, Mat &mx2, Mat &my2)
{
	sz = size;
	windowSize = wsz;
	windowLength = 2 * wsz + 1;
	windowArea = windowLength * windowLength;
	rows = size.height;
	cols = size.width;
	totalPixel = rows * cols;
	numDisparity = numDisp;

	// allocate memory for internal production
	hipMalloc(&d_difference, numDisparity * totalPixel * sizeof(uchar));

	hipMalloc(&d_left, totalPixel * sizeof(uchar3));
	hipMalloc(&d_right, totalPixel * sizeof(uchar3));
	hipMalloc(&d_left_remapped, totalPixel * sizeof(uchar));
	hipMalloc(&d_right_remapped, totalPixel * sizeof(uchar));
	hipMalloc(&d_left_cvted, totalPixel * sizeof(uchar));
	hipMalloc(&d_right_cvted, totalPixel * sizeof(uchar));

	// allocate memory for result
	hipMalloc(&d_disparity, totalPixel * sizeof(uchar));
	hipMalloc(&d_filtered_disp, totalPixel * sizeof(uchar));
	h_disparity = new uchar[totalPixel];

	// allocate memory for calib data
	hipMalloc(&d_x1, totalPixel * sizeof(float));
	hipMalloc(&d_y1, totalPixel * sizeof(float));
	hipMalloc(&d_x2, totalPixel * sizeof(float));
	hipMalloc(&d_y2, totalPixel * sizeof(float));

	// copy data to GPU
	hipMemcpy(d_x1, mx1.data, totalPixel * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y1, my1.data, totalPixel * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_x2, mx2.data, totalPixel * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y2, my2.data, totalPixel * sizeof(float), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	filter = guidedFilterGPU(rows, cols, 2, 2, 255 * 255 * 0.02 * 0.02);

	// test
	hipMalloc(&ftemp1, totalPixel * sizeof(float));
	hipMalloc(&ftemp2, totalPixel * sizeof(float));
	hipMalloc(&fresult, totalPixel * sizeof(float));
	h_fresult = new float[totalPixel];
	hipMalloc(&utemp1, totalPixel * sizeof(uchar));
	hipMalloc(&utemp2, totalPixel * sizeof(uchar));
	hipMalloc(&uresult, totalPixel * sizeof(uchar));
	h_uresult = new uchar[totalPixel];
}

Device::~Device()
{
}

// proxy function
void Device::blockMatching_gpu(Mat &h_left, Mat &h_right, Mat &h_disparity, int SADWindowSize, int searchRange)
{
	uchar *d_disparity, *d_left, *d_right, *d_difference, *d_sad_data;
	uchar *h_disparity_data;
	Point3i *h_relativeLocation, *d_relativeLocation;
	int cols = h_left.cols;
	int rows = h_left.rows;
	int total = cols * rows;
	int windowLength = 2 * SADWindowSize + 1;
	int windowArea = windowLength * windowLength;

	// malloc data
	h_disparity_data = new uchar[total];
	h_relativeLocation = new Point3i[windowArea];
	hipMalloc(&d_sad_data, total * searchRange * sizeof(uchar));
	hipMalloc(&d_relativeLocation, windowArea * sizeof(Point3i));
	hipMalloc(&d_left, total * sizeof(uchar));
	hipMalloc(&d_right, total * sizeof(uchar));
	hipMalloc(&d_disparity, total * sizeof(uchar));
	hipMemset(d_disparity, 0, total * sizeof(uchar));
	hipMalloc(&d_difference, searchRange * total * sizeof(uchar));
	hipMemset(d_difference, 0, searchRange * total * sizeof(uchar));

	// clock_t start, end;
	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// 1. upload data to GPU memory
	/**************************************************************************************/
	/**************************************************************************************/
	hipEventRecord(start, 0);
	hipMemcpy(d_left, h_left.data, total * sizeof(uchar), hipMemcpyHostToDevice);
	hipMemcpy(d_right, h_right.data, total * sizeof(uchar), hipMemcpyHostToDevice);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	cout << "upload data : " << elapsedTime << endl;
	/**************************************************************************************/
	/**************************************************************************************/


	// 2. pre-calculate difference
	/**************************************************************************************/
	/**************************************************************************************/
	hipEventRecord(start, 0);
	// naive pre-calculation
	// kernelPreCal << <1, searchRange >> >(d_left, d_right, d_difference, cols, rows, total);

	// optimized pre-calculation
	dim3 block = dim3(32, 32, 1);
	dim3 grid = dim3(8, 10, searchRange);
	kernelPreCal_V2 << <grid, block >> >(d_left, d_right, d_difference, cols, rows, total);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	cout << "pre calculation : " << elapsedTime << endl;

	// DEBUG : compare the difference to CPU's result
	// uchar *result = new uchar[searchRange * total];
	// hipMemcpy(result, d_difference, searchRange * total * sizeof(uchar), hipMemcpyDeviceToHost);
	// compareDiff(h_left, h_right, result, SADWindowSize, searchRange, total);
	/**************************************************************************************/
	/**************************************************************************************/


	// 3. find correspondance
	/**************************************************************************************/
	/**************************************************************************************/
	hipEventRecord(start, 0);
	// naive method to find correspondance
	kernelFindCorr << <rows, cols >> >(d_difference, d_disparity, cols, rows, windowArea, searchRange, total, windowLength, SADWindowSize);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	cout << "find corr : " << elapsedTime << endl;

	// optimized method
	/*
	dim3 resolution = dim3(rows, cols, 1);
	start = clock();
	kernelFindAllSAD << <grid, block >> >(d_left, d_right, d_difference, d_relativeLocation, d_sad_data, cols, rows, windowArea, searchRange, total, windowLength, SADWindowSize);
	hipDeviceSynchronize();
	// uchar *h_sad_data = new uchar[total * searchRange];
	// hipMemcpy(h_sad_data, d_sad_data, total * searchRange * sizeof(uchar), hipMemcpyDeviceToHost);
	// compareSAD(h_left, h_right, h_sad_data, SADWindowSize, searchRange, cols, rows);
	// getAllSAD(h_left, h_right, h_sad_data, SADWindowSize, searchRange);
	// hipMemcpy(d_sad_data, h_sad_data, total * searchRange * sizeof(uchar), hipMemcpyHostToDevice);
	end = clock();
	cout << "find corr V2 : " << (double)(end - start) / CLOCKS_PER_SEC << endl;


	start = clock();
	kernelFindMinSAD << <resolution, searchRange >> >(d_sad_data, d_disparity, cols, searchRange);
	hipDeviceSynchronize();
	end = clock();
	cout << "find min : " << (double)(end - start) / CLOCKS_PER_SEC << endl;
	*/
	/**************************************************************************************/
	/**************************************************************************************/


	// 4. download data
	/**************************************************************************************/
	/**************************************************************************************/
	hipEventRecord(start, 0);
	hipMemcpy(h_disparity_data, d_disparity, total * sizeof(uchar), hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	cout << "download data : " << elapsedTime << endl;
	/**************************************************************************************/
	/**************************************************************************************/

	// DEBUG : compare the disparity to CPU's result
	// compareDisp(h_left, h_right, h_disparity_data, SADWindowSize, searchRange, cols, rows);

	// 5. return data
	h_disparity = Mat(rows, cols, CV_8UC1, h_disparity_data);
}

void Device::remap_gpu(Mat &left, Mat &right, Mat &mapX1, Mat &mapY1, Mat &mapX2, Mat &mapY2, int rows, int cols, int total, uchar *result)
{
	uchar *d_left_gpu_data, *d_right_gpu_data, *d_left, *d_right;
	float *d_mapx1, *d_mapx2, *d_mapy1, *d_mapy2;
	hipMalloc(&d_left_gpu_data, total * sizeof(uchar));
	hipMalloc(&d_right_gpu_data, total * sizeof(uchar));
	hipMalloc(&d_left, total * sizeof(uchar));
	hipMalloc(&d_right, total * sizeof(uchar));
	hipMalloc(&d_mapx1, total * sizeof(float));
	hipMalloc(&d_mapx2, total * sizeof(float));
	hipMalloc(&d_mapy1, total * sizeof(float));
	hipMalloc(&d_mapy2, total * sizeof(float));

	hipMemcpy(d_mapx1, mapX1.data, total * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_mapx2, mapX2.data, total * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_mapy1, mapY1.data, total * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_mapy2, mapY2.data, total * sizeof(float), hipMemcpyHostToDevice);

	hipMemcpy(d_left, left.data, total * sizeof(uchar), hipMemcpyHostToDevice);
	hipMemcpy(d_right, right.data, total * sizeof(uchar), hipMemcpyHostToDevice);

	
	hipEvent_t start, stop;
	float elapsedTime;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	kernelRemap << <rows, cols >> >(d_left, d_left_gpu_data, d_mapx1, d_mapy1, rows, cols);
	kernelRemap << <rows, cols >> >(d_right, d_right_gpu_data, d_mapx2, d_mapy2, rows, cols);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	cout << "GPU Remap : " << elapsedTime << endl;

	hipMemcpy(result, d_left_gpu_data, total * sizeof(uchar), hipMemcpyDeviceToHost);
}

void Device::cvtColor_gpu(uchar3 *src, uchar *dst, int rows, int cols)
{
	uchar3 *d_src;
	uchar *d_dst;
	int total = rows * cols;
	hipEvent_t start, end;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&end);

	hipMalloc(&d_src, total * sizeof(uchar3));
	hipMalloc(&d_dst, total * sizeof(uchar));
	hipMemcpy(d_src, src, total * sizeof(uchar3), hipMemcpyHostToDevice);

	hipEventRecord(start, 0);
	for (size_t i = 0; i < 1000; i++)
		kernelCvtColor << <rows, cols >> >(d_src, d_dst, rows, cols);
	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&time, start, end);
	cout << "GPU cvtColor : " << time << endl;

	hipMemcpy(dst, d_dst, total * sizeof(uchar), hipMemcpyDeviceToHost);
}

void Device::pipeline(Mat &left, Mat &right)
{
	// resize
	resize(left, left, sz);
	resize(right, right, sz);
	imshow("Left", left);
	imshow("Right", right);

	// upload data
	hipMemcpy(d_left, left.data, totalPixel * sizeof(uchar3), hipMemcpyHostToDevice);
	hipMemcpy(d_right, right.data, totalPixel * sizeof(uchar3), hipMemcpyHostToDevice);

	// convert color
	kernelCvtColor << <rows, cols >> >(d_left, d_left_cvted, rows, cols);
	kernelCvtColor << <rows, cols >> >(d_right, d_right_cvted, rows, cols);

	// remap
	kernelRemap << <rows, cols >> >(d_left_cvted, d_left_remapped, d_x1, d_y1, rows, cols);
	kernelRemap << <rows, cols >> >(d_right_cvted, d_right_remapped, d_x2, d_y2, rows, cols);

	// stereo matching
	dim3 block = dim3(24, 32, 1);
	dim3 grid = dim3(10, 10, numDisparity);
	kernelPreCal_V2 << <grid, block >> >(d_left_remapped, d_right_remapped, d_difference, cols, rows, totalPixel);
	kernelFindCorr << <rows, cols >> >(d_difference, d_disparity, cols, rows, windowArea, numDisparity, totalPixel, windowLength, windowSize);

	// download data(no filter)
	//hipMemcpy(h_disparity, d_disparity, totalPixel * sizeof(uchar), hipMemcpyDeviceToHost);
	//imshow("Disp", Mat(rows, cols, CV_8UC1, h_disparity));

	// guided filter 
	filter.filter(d_disparity, d_disparity, d_filtered_disp);
	hipMemcpy(h_disparity, d_filtered_disp, totalPixel * sizeof(uchar), hipMemcpyDeviceToHost);
	imshow("Disp", Mat(rows, cols, CV_8UC1, h_disparity));

	// test
	// gpu gilter
	//filter.filter(d_disparity, d_disparity, d_filtered_disp);
	//// cpu filter 
	//hipMemcpy(h_disparity, d_disparity, totalPixel * sizeof(uchar), hipMemcpyDeviceToHost);
	//Mat cpu_result = Mat(rows, cols, CV_8UC1, h_disparity);
	//GuidedFilter gf = GuidedFilter(cpu_result, 2, 0.02 * 0.02 * 255 * 255);
	//Mat dd = gf.filter(cpu_result);
	//imshow("CPU", dd);
	//// compare the result 
	//hipMemcpy(h_fresult, filter.result_float, totalPixel * sizeof(float), hipMemcpyDeviceToHost);
	//imshow("GPU float", Mat(rows, cols, CV_32FC1, h_fresult));

	//hipMemcpy(h_uresult, d_filtered_disp, totalPixel * sizeof(uchar), hipMemcpyDeviceToHost);
	//imshow("GPU uchar", Mat(rows, cols, CV_8UC1, h_uresult));
	//for (size_t i = 0; i < rows; i++)
	//{
	//	for (size_t j = 0; j < cols; j++)
	//	{
	//		int index = i * cols + j;
	//		if (h_fresult[index] != gf.impl_->getMat("result").ptr<float>(i)[j])
	//		{
	//			cout << "Diff : " << '[' << i << ',' << j << "]\t" << h_fresult[index] << '\t' << (float)gf.impl_->getMat("result").ptr<float>(i)[j] << endl;
	//		}
	//		else
	//		{
	//			cout << "Same : " << '[' << i << ',' << j << "]\t" << h_fresult[index] << '\t' << (float)gf.impl_->getMat("result").ptr<float>(i)[j] << endl;
	//		}
	//	}
	//}
	//imshow("convert", Mat(rows, cols, CV_32FC1, h_fresult));
	//kernelBoxFilter<<<rows, cols>>>(temp1, result, filter.r, filter.c, rows, cols);
	//hipMemcpy(h_result, result, totalPixel * sizeof(float), hipMemcpyDeviceToHost);
	//imshow("BoxFilter", Mat(rows, cols, CV_32FC1, h_result));
}

void Device::pipeline2(Mat &left, Mat &right)
{
	// resize
	resize(left, left, sz);
	resize(right, right, sz);
	imshow("Left", left);
	imshow("Right", right);

	// upload data
	hipMemcpy(d_left, left.data, totalPixel * sizeof(uchar3), hipMemcpyHostToDevice);
	hipMemcpy(d_right, right.data, totalPixel * sizeof(uchar3), hipMemcpyHostToDevice);

	// convert color
	kernelCvtColor << <rows, cols >> >(d_left, d_left_cvted, rows, cols);
	kernelCvtColor << <rows, cols >> >(d_right, d_right_cvted, rows, cols);

	// remap
	kernelRemap << <rows, cols >> >(d_left_cvted, d_left_remapped, d_x1, d_y1, rows, cols);
	kernelRemap << <rows, cols >> >(d_right_cvted, d_right_remapped, d_x2, d_y2, rows, cols);

	// stereo matching
	//dim3 block = dim3(24, 32, 1);
	//dim3 grid = dim3(10, 10, numDisparity);
	//kernelPreCal_V2 << <grid, block >> >(d_left_remapped, d_right_remapped, d_difference, cols, rows, totalPixel);
	kernelFindCorrNonPreCal << <rows, cols >> >(d_left_remapped, d_right_remapped, d_disparity, cols, rows, windowArea, numDisparity, totalPixel, windowLength, windowSize);

	// download data
	hipMemcpy(h_disparity, d_disparity, totalPixel * sizeof(uchar), hipMemcpyDeviceToHost);
	imshow("Disp", Mat(rows, cols, CV_8UC1, h_disparity));
}