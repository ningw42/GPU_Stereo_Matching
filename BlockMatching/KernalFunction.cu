#include "hip/hip_runtime.h"
#include "KernalFunction.cuh"

__global__ void kernalBoxFilter(float *src, float *dst, int r, int c, int rows, int cols)
{
	int row = blockIdx.x, col = threadIdx.x;
	int index = row * blockDim.x + col;
	float sum = 0;
	for (size_t currRow = MAX(row - r, 0); currRow <= MIN(rows, row + r); currRow++)
	{
		for (size_t currCol = MAX(col - c, 0); currCol <= MIN(cols, col + c); currCol++)
		{
			sum += src[currRow * cols + currCol];
		}
	}
	//float temp = sum / (r * c);
	dst[index] = sum / (r * c);
}

__global__ void kernalMul(float *first, float *second, float *result)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	result[index] = first[index] * second[index];
}

__global__ void kernalDivide(float *first, float *second, float *result)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	result[index] = first[index] / second[index];
}

__global__ void kernalSub(float *first, float *second, float *result)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	result[index] = first[index] - second[index];
}

__global__ void kernalAddEle(float *first, float e, float *result)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	result[index] = first[index] + e;
}

__global__ void kernalAdd(float *first, float *second, float *result)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	result[index] = first[index] + second[index];
}

__global__ void kernalConvertToFloat(uchar *src, float *dst)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	dst[index] = uchar2float(src[index]);
}

__global__ void kernalConvertToUchar(float *src, uchar *dst)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	dst[index] = float2uchar(src[index]);
}

__device__ __forceinline__ float uchar2float(uchar a)
{
	return (float)a;
}

__device__ __forceinline__ uchar float2uchar(float a)
{
	unsigned int res = 0;
	asm("cvt.rni.sat.u8.f32 %0, %1;" : "=r"(res) : "f"(a));
	return res;
}


__global__ void kernalPreCal(uchar *left, uchar *right, uchar *difference, int numberOfCols, int numberOfRows, int total)
{
	int index = threadIdx.x;
	int th = index * total;

	for (size_t i = 0; i < total; i++)
	{
		int c = i % numberOfCols - index;
		if (c < 0) continue;
		difference[i + th] = (uchar)std::abs(left[i] - right[i - index]);
	}
}

__global__ void kernalPreCal_V2(uchar *left, uchar *right, uchar *difference, int numberOfCols, int numberOfRows, int total)
{
	int colIndex = blockIdx.y * blockDim.y + threadIdx.y;
	int rowIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int frameBias = rowIndex * numberOfCols + colIndex;
	int frameIndex = blockIdx.z;
	int index = frameIndex * total + frameBias;

	// calculate difference only if two pixels are at the same line 
	int refCol = colIndex - frameIndex;
	if (refCol >= 0)
	{

		//difference[index] = (uchar)__usad(left[frameBias] - right[frameBias - frameIndex]);
		difference[index] = (uchar)std::abs(left[frameBias] - right[frameBias - frameIndex]);
	}
}

__global__ void kernalFindCorr(uchar *difference, uchar *disparity, int numberOfCols, int numberOfRows, int windowArea, int searchRange, int total, int windowsLength, int SADWinwdowSize)
{
	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int currentMinSAD = 50 * windowArea;
	int matchedPosDisp = 0;
	int col = threadIndex % numberOfCols;
	int row = threadIndex / numberOfCols;
	int th = 0;

	for (int _search = 0; _search < searchRange; _search++, th += total) {
		if (col + _search > numberOfCols) break;
		int SAD = 0;
		// calculate the SAD of the current disparity
		for (int i = -SADWinwdowSize; i <= SADWinwdowSize; i++)
		{
			for (int j = -SADWinwdowSize; j <= SADWinwdowSize; j++)
			{
				int _col = col + j;
				if (_col >= numberOfCols || _col < 0) continue;
				int _row = row + i;
				if (_row >= numberOfRows || _row < 0) continue;
				SAD += difference[th + threadIndex + numberOfCols * i + j];
			}
		}
		if (SAD < currentMinSAD) {
			matchedPosDisp = _search;
			currentMinSAD = SAD;
		}
	}

	disparity[threadIndex] = matchedPosDisp;
}

__global__ void kernalFindCorrNonPreCal(uchar *left, uchar *right, uchar *disparity, int numberOfCols, int numberOfRows, int windowArea, int searchRange, int total, int windowsLength, int SADWinwdowSize)
{
	// grid and block should be <rows, cols> respectively
	int col = threadIdx.x;
	int row = blockIdx.x;
	int threadIndex = row * blockDim.x + col;
	int currentMinSAD = 20 * windowArea;
	int matchedPosDisp = 0;

	for (int _search = 0; _search < searchRange; _search++) {
		if (col + _search > numberOfCols) break;
		int SAD = 0;
		// calculate the SAD of the current disparity
		for (int i = -SADWinwdowSize; i <= SADWinwdowSize; i++)
		{
			for (int j = -SADWinwdowSize; j <= SADWinwdowSize; j++)
			{
				int _col = col + j;
				if (_col >= numberOfCols || _col < 0) continue;
				int _row = row + i;
				if (_row >= numberOfRows || _row < 0) continue;
				int base = threadIndex + numberOfCols * i + j;
				SAD += (uchar)std::abs(left[base + _search] - right[base]);
			}
		}
		if (SAD < currentMinSAD) {
			matchedPosDisp = _search;
			currentMinSAD = SAD;
		}
	}

	disparity[threadIndex] = matchedPosDisp;
}

// a pair of function to get the matched position
__global__ void kernalFindAllSAD(uchar *left, uchar *right, uchar *difference, uchar *SAD_data, int numberOfCols, int numberOfRows, int windowArea, int searchRange, int total, int windowLength, int SADWinwdowSize)
{
	int colIndex = blockIdx.y * blockDim.y + threadIdx.y;
	int rowIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int frameBias = rowIndex * numberOfCols + colIndex;
	int frameIndex = blockIdx.z;
	int index = frameIndex * total + frameBias;

	if (colIndex + frameIndex > numberOfCols)
	{
		SAD_data[frameBias * searchRange + frameIndex] = 255;
		return;
	}

	int SAD = 0;
	int currCol, currRow;

	for (int i = -SADWinwdowSize; i <= SADWinwdowSize; i++)
	{
		for (int j = -SADWinwdowSize; j <= SADWinwdowSize; j++)
		{
			currCol = colIndex + j;
			if (currCol >= numberOfCols || currCol < 0) continue;
			currRow = rowIndex + i;
			if (currRow >= numberOfRows || currRow < 0) continue;
			SAD += difference[index + i * numberOfCols + j];
		}
	}

	SAD_data[frameBias * searchRange + frameIndex] = SAD;
}

__global__ void kernalFindMinSAD(uchar *SAD_data, uchar *disparity, int numberOfCols, int searchRange)
{
	// TO-DO: return the original index of the min element as the matched position
	int step = threadIdx.x;
	int frameBias = blockIdx.x * numberOfCols + blockIdx.y;
	int base = searchRange * frameBias;
	int matchedPos = 0;

	int index = step + base;
	for (size_t i = blockDim.x / 2; i > 0; i = i >> 1)
	{
		if (step < i)
		{
			if (SAD_data[index] < SAD_data[index + i])
				SAD_data[index] = SAD_data[index];
			else
				SAD_data[index] = SAD_data[index + i];
			// SAD_data[index] = min(SAD_data[index], SAD_data[index + i]);
		}
		__syncthreads();
	}

	if (step == 0)
	{
		disparity[frameBias] = matchedPos;
	}
}

__global__ void kernalRemap(uchar *src, uchar *dst, float *mapx, float *mapy, int rows, int cols)
{
	int index = blockIdx.x * cols + threadIdx.x;

	const float xcoo = mapx[index];
	const float ycoo = mapy[index];
	dst[index] = float2uchar(BilinearInterpolation(src, rows, cols, ycoo, xcoo));
}

__global__ void kernalCvtColor(uchar3 *src, uchar *dst, int rows, int cols)
{
	int index = blockIdx.x * cols + threadIdx.x;

	uchar3 rgb = src[index];
	float channelSum = .299f * rgb.x + .587f * rgb.y + .114f * rgb.z;
	dst[index] = float2uchar(channelSum);
}

//__device__ __forceinline__ uchar float2uchar(float a)
//{
//	unsigned int res = 0;
//	asm("cvt.rni.sat.u8.f32 %0, %1;" : "=r"(res) : "f"(a));
//	return res;
//}

__device__ float BilinearInterpolation(uchar *src, int rows, int cols, float x, float y)
{
	int x1 = floorf(x), y1 = floorf(y), x2 = x1 + 1, y2 = y1 + 1;
	if (x1 < 0 || x2 >= rows || y1 < 0 || y2 >= cols) {
		return 0;
	}

	int baseIndex = x1 * cols + y1;
	uchar Q11 = src[baseIndex], Q12 = src[baseIndex + 1], Q21 = src[baseIndex + cols], Q22 = src[baseIndex + cols + 1];

	float left = (x2 - x) * Q11 + (x - x1) * Q21;
	float right = (x2 - x) * Q12 + (x - x1) * Q22;

	float result = (y2 - y) * left + (y - y1) * right;
	return result;
}
